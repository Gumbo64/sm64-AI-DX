// vector_add.cu

#include <iostream>
#include <hip/hip_runtime.h>

#define N 1024 // Size of the vectors

// CUDA kernel to add two vectors
__global__ void vectorAdd(const float *a, const float *b, float *c, int n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

int main() {
    float *h_a, *h_b, *h_c; // Host vectors
    float *d_a, *d_b, *d_c; // Device vectors

    size_t size = N * sizeof(float);

    // Allocate host memory
    h_a = (float *)malloc(size);
    h_b = (float *)malloc(size);
    h_c = (float *)malloc(size);

    // Initialize host vectors
    for (int i = 0; i < N; i++) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }

    // Allocate device memory
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    // Copy host memory to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // Launch kernel with 1 block of N threads
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N);

    // Copy result back to host
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    // Print some results
    for (int i = 0; i < 10; i++) { // Print first 10 elements
        std::cout << h_a[i] << " + " << h_b[i] << " = " << h_c[i] << std::endl;
    }

    // Clean up
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
